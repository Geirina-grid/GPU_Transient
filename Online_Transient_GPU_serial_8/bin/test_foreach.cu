#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

struct sum_Functor {
    int *sum;
    sum_Functor(int *s){sum = s;}
    __host__ __device__
    void operator()(int i)
    {
        *sum+=i;
        printf("In functor: i %d sum %d\n",i,*sum);
    }

};

int main(){

    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first+10;
    int sum = 0;
    sum_Functor sf(&sum);
    printf("After constructor: value is %d\n", *(sf.sum));
    for(int i=0;i<5;i++){
        sf(i);
    }

    printf("Initiating for_each call - current value %d\n", (*(sf.sum)));
    thrust::for_each(first,last,sf);

    hipDeviceSynchronize();
    printf("After for_each: value is %d\n",*(sf.sum));
}
